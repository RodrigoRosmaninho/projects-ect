#include "hip/hip_runtime.h"
//
// António Rui Borges
//
// ACA 2020/2021
//
// Reference implementation
//

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "common.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// program configuration
//

#ifndef N
# define N  (1 << 16)
#endif

static void cyclicCircConv_cpu_kernel (float *x, float *y, float *xy, unsigned int nSamp);
__global__ static void cyclicCircConv_cuda_kernel (float *x, float *y, float *xy, unsigned int nSamp);
static double get_delta_time(void);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Main program
//

int main (int argc, char **argv)
{
  printf("%s Starting...\n", argv[0]);

  // set up device
  int dev = 0;
  int i;

  hipDeviceProp_t deviceProp;
  CHECK (hipGetDeviceProperties (&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK (hipSetDevice (dev));

  // create memory areas in host and device memory where the signals and their ccyclic circular convolution will be stored
  float *host_x, *host_y, *host_xy;

  host_x = (float *) malloc (N * sizeof (float));
  host_y = (float *) malloc (N * sizeof (float));
  host_xy = (float *) malloc (N * sizeof (float));

  // initialize signal datacyclicCircConv_cuda_kernel
  (void) get_delta_time ();
  srand(0xACA2020);
  for (i = 0; i < N; i++)
  { host_x[i] = (float) ((double) rand () / RAND_MAX - 0.5);
    host_y[i] = (float) ((double) rand () / RAND_MAX - 0.5);
  }
  printf ("The initialization of host data took %.3e seconds\n",get_delta_time ());

  // create memory areas in device memory and copy the host data to the device memory
  float *device_x, *device_y, *device_xy;;

  (void) get_delta_time ();
  CHECK (hipMalloc ((void **) &device_x, N * sizeof (float)));
  CHECK (hipMalloc ((void **) &device_y, N * sizeof (float)));
  CHECK (hipMalloc ((void **) &device_xy, N * sizeof (float)));
  CHECK (hipMemcpy (device_x, host_x, N * sizeof (float), hipMemcpyHostToDevice));
  CHECK (hipMemcpy (device_y, host_y, N * sizeof (float), hipMemcpyHostToDevice));
  printf ("The creation and transfer of %ld bytes from the host to the device took %.3e seconds\n",
          (long) N * sizeof (float), get_delta_time ());

  // run the computational kernel in the GPU
  // as an example, N thread blocks are launched where each thread block deals solely with one convolution point
  unsigned int gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ;

  blockDimX = 1 << 8;                                                    // optimize!
  blockDimY = 1 << 0;                                                    // optimize!
  blockDimZ = 1 << 0;                                                    // optimize!
  gridDimX = 1 << 8;                                                     // optimize!
  gridDimY = 1 << 0;                                                     // optimize!
  gridDimZ = 1 << 0;                                                     // optimize!
  if ((blockDimX * blockDimY * blockDimZ * gridDimX * gridDimY * gridDimZ) != N)
     { fprintf (stderr,"Wrong launch configuration!\n");
       exit (1);
     }

  dim3 grid (gridDimX, gridDimY, gridDimZ);
  dim3 block (blockDimX, blockDimY, blockDimZ);

  (void) get_delta_time ();
  cyclicCircConv_cuda_kernel <<<grid, block>>> (device_x, device_y, device_xy, (unsigned int) N);
  CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
  CHECK (hipGetLastError ());                                 // check for kernel errors
  printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

  // copy kernel result back to host side
  float *modified_device_xy;

  modified_device_xy = (float *) malloc (N * sizeof (float));
  CHECK (hipMemcpy (modified_device_xy, device_xy, N * sizeof (float), hipMemcpyDeviceToHost));
  printf ("The transfer of %ld bytes from the device to the host took %.3e seconds\n",
          (long) N * sizeof (float), get_delta_time ());

  // free device global memory
  CHECK (hipFree (device_x));
  CHECK (hipFree (device_y));
  CHECK (hipFree (device_xy));

  // reset device
  CHECK (hipDeviceReset ());

  // run the computational kernel in the CPU
  (void) get_delta_time ();
  cyclicCircConv_cpu_kernel (host_x, host_y, host_xy, (unsigned int) N);
  printf("The cpu kernel took %.3e seconds to run (single core)\n", get_delta_time ());

  // compare
  for(i = 0; i < N; i++)
    if ((fabsf (host_xy[i]) < 1e-3) && (fabsf (modified_device_xy[i]) > 1.01e-3))
       { printf ("Mismatch in sample point %d: cpu %.3e - gpu %.3e\n", i, host_xy[i], modified_device_xy[i]);
         exit(1);
       }
       else if (fabsf ((host_xy[i] - modified_device_xy[i]) / host_xy[i]) >= 5e-2)
       { printf ("Mismatch in sample point %d: cpu %.3e - gpu %.3e\n", i, host_xy[i], modified_device_xy[i]);
         exit(1);
       }
  printf ("All is well!\n");

  // free host memory
  free (host_x);
  free (host_y);
  free (host_xy);
  free (modified_device_xy);

  return 0;
}

static void cyclicCircConv_cpu_kernel (float *x, float *y, float *xy, unsigned int nSamp)
{
  unsigned int i, k;
  float tmp;

  for (i = 0; i < nSamp; i++)
  { tmp = 0.0;
    for (k = 0; k < nSamp; k++)
      tmp += x[k] * y[(i + k) % nSamp];
    xy[i] += tmp;
  }
}

__global__ static void cyclicCircConv_cuda_kernel (float *xx, float *yy, float *xxyy, unsigned int nSamp)
{
  unsigned int k, x, y, z, idx;
  float tmp;

  // compute the thread number
  x = (unsigned int) threadIdx.x + (unsigned int) blockDim.x * (unsigned int) blockIdx.x;
  y = (unsigned int) threadIdx.y + (unsigned int) blockDim.y * (unsigned int) blockIdx.y;
  z = (unsigned int) threadIdx.z + (unsigned int) blockDim.z * (unsigned int) blockIdx.z;
  idx = (unsigned int) blockDim.y * (unsigned int) gridDim.y * (unsigned int) blockDim.x * (unsigned int) gridDim.x * z +
        (unsigned int) blockDim.x * (unsigned int) gridDim.x * y + x;
  if (idx >= nSamp)
     { printf ("Out of the data array: %u!\n", idx);
       return;                                             // safety precaution
     }

  tmp = 0.0;
  for (k = 0; k < nSamp; k++)
    tmp += xx[k] * yy[(idx + k) % nSamp];
  xxyy[idx] += tmp;
}

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
